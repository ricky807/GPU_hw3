#include "hip/hip_runtime.h"
// 
//#ifndef _SIMPLE_KERNEL_H_
//#define _SIMPLE_KERNEL_H_

#include <stdio.h>
#include "kernel.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k0( float* g_dataA, float* g_dataB, int pitch, int width ) 
{

  // global thread(data) row index 
  unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
  i = i + 1; //because the edge of the data is not processed
  
  // global thread(data) column index
  unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
  j = j + 1; //because the edge of the data is not processed  

  // check the boundary
  if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) return;

  g_dataB[i * pitch + j] = (
                              0.2f * g_dataA[i * pitch + j] +               //itself
                              0.1f * g_dataA[(i-1) * pitch +  j   ] +       //N
                              0.1f * g_dataA[(i-1) * pitch + (j+1)] +       //NE
                              0.1f * g_dataA[ i    * pitch + (j+1)] +       //E
                              0.1f * g_dataA[(i+1) * pitch + (j+1)] +       //SE
                              0.1f * g_dataA[(i+1) * pitch +  j   ] +       //S
                              0.1f * g_dataA[(i+1) * pitch + (j-1)] +       //SW
                              0.1f * g_dataA[ i    * pitch + (j-1)] +       //W
                              0.1f * g_dataA[(i-1) * pitch + (j-1)]         //NW
                           ) * 0.95f;
}

// #endif // #ifndef _SIMPLE_KERNEL_H_
