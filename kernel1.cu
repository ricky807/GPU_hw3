#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
    extern __shared__ float s_data[];
    // TODO, implement this kernel below

    // global thread(data) row index 
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    i = i + 1;
  
    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int blocksize = blockIdx.x + 2;
    j = j + 1;

    int shared_i_index = threadIdx.y;
    int shared_j_index = threadIdx.x;
    if(!(i >= width - 1|| j >= width - 1 || i < 1 || j < 1))
    {
        
        s_data[shared_i_index-1 * blocksize + shared_j_index] = g_dataA[i-1 * floatpitch + j];
        s_data[shared_i_index * blocksize + shared_j_index] = g_dataA[i * floatpitch + j];
        s_data[shared_i_index+1 * blocksize + shared_j_index] = g_dataA[i+1 * floatpitch + j];
        
        if(shared_j_index == 1)
        {
            s_data[shared_i_index-1 * blocksize + shared_j_index-1] = g_dataA[i * floatpitch + j-1];
            s_data[shared_i_index * blocksize + shared_j_index-1] = g_dataA[i+1 * floatpitch + j-1];
            s_data[shared_i_index+1 * blocksize + shared_j_index-1] = g_dataA[i+2 * floatpitch + j-1];
        }
        if(shared_j_index == blockDim.x -1)
        {
            s_data[shared_i_index-1 * blocksize + shared_j_index+1] = g_dataA[i * floatpitch + j+1];
            s_data[shared_i_index * blocksize + shared_j_index+1] = g_dataA[i+1 * floatpitch + j+1];
            s_data[shared_i_index+1 * blocksize + shared_j_index+1] = g_dataA[i+2 * floatpitch + j+1];
        }
        
    }

    __syncthreads();
    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) return;
    
    g_dataB[i * floatpitch + j] = (
                                0.2f * s_data[i * blocksize + j] +               //itself
                                0.1f * s_data[(i-1) * blocksize +  j   ] +       //N
                                0.1f * s_data[(i-1) * blocksize + (j+1)] +       //NE
                                0.1f * s_data[ i    * blocksize + (j+1)] +       //E
                                0.1f * s_data[(i+1) * blocksize + (j+1)] +       //SE
                                0.1f * s_data[(i+1) * blocksize +  j   ] +       //S
                                0.1f * s_data[(i+1) * blocksize + (j-1)] +       //SW
                                0.1f * s_data[ i    * blocksize + (j-1)] +       //W
                                0.1f * s_data[(i-1) * blocksize + (j-1)]         //NW
                            ) * 0.95f;
    
    
}

